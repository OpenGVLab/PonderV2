#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "subtraction_cuda_kernel.h"


__global__ void subtraction_forward_cuda_kernel(int n, int nsample, int c, const float *input1, const float *input2, const int *idx, float *output) {
    // input: input1: (n, c), input2: (n, c), idx: (n, nsample), output: (n, nsample, c)
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n * nsample * c) return;
    const int c_idx = index % c;
    const int nsample_idx = (index / c) % nsample;
    const int n_idx = index / nsample / c;
    const int idx_idx = n_idx * nsample + nsample_idx;
    const int input1_idx = n_idx * c + c_idx;
    const int input2_idx = idx[idx_idx] * c + c_idx;
    output[index] = input1[input1_idx] - input2[input2_idx];
}

__global__ void subtraction_backward_cuda_kernel(int n, int nsample, int c, const int *idx, const float *grad_output, float *grad_input1, float *grad_input2) {
    // input: grad_output: (n, nsample, c), output: grad_input1: (n, c), grad_input2: (n, c)
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n * nsample * c) return;
    const int c_idx = index % c;
    const int nsample_idx = (index / c) % nsample;
    const int n_idx = index / nsample / c;
    const int idx_idx = n_idx * nsample + nsample_idx;
    const int input1_idx = n_idx * c + c_idx;
    const int input2_idx = idx[idx_idx] * c + c_idx;
    atomicAdd(grad_input1 + input1_idx, grad_output[index]);
    atomicAdd(grad_input2 + input2_idx, -grad_output[index]);
}

void subtraction_forward_cuda_launcher(int n, int nsample, int c, const float *input1, const float *input2, const int *idx, float *output) {
    // input: input1: (n, c), input2: (n, c), idx: (n, nsample), output: (n, nsample, c)
    dim3 blocks(DIVUP(n * nsample * c, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);
    subtraction_forward_cuda_kernel<<<blocks, threads, 0>>>(n, nsample, c, input1, input2, idx, output);
}

void subtraction_backward_cuda_launcher(int n, int nsample, int c, const int *idx, const float *grad_output, float *grad_input1, float *grad_input2) {  
    // input: grad_output: (n, nsample, c), output: grad_input1: (n, c), grad_input2: (n, c)
    dim3 blocks(DIVUP(n * nsample * c, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);
    subtraction_backward_cuda_kernel<<<blocks, threads, 0>>>(n, nsample, c, idx, grad_output, grad_input1, grad_input2);
}
