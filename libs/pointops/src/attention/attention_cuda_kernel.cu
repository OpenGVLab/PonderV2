#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "attention_cuda_kernel.h"


/*
Kernels
*/

__global__ void attention_relation_step_forward_cuda_kernel(int m, int g, int c,
                                                            const float *query, const float *key, const float *weight,
                                                            const int *index_target, const int *index_refer,
                                                            float *output)
{
    int r_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int g_idx = blockIdx.y;
    int c_idx = blockIdx.z;

    if (r_idx >= m || g_idx >= g || c_idx >= c) return;
    int q_idx = index_target[r_idx] * g * c + g_idx * c + c_idx;
    int k_idx = index_refer[r_idx] * g * c + g_idx * c + c_idx;

    float r = query[q_idx] * key[k_idx] * weight[c_idx];
    atomicAdd(output + r_idx * g + g_idx, r);
}

__global__ void attention_relation_step_backward_cuda_kernel(int m, int g, int c,
                                                             const float *query, float *grad_query,
                                                             const float *key, float *grad_key,
                                                             const float *weight, float *grad_weight,
                                                             const int *index_target, const int *index_refer,
                                                             const float *grad_output)
{
    int r_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int g_idx = blockIdx.y;
    int c_idx = blockIdx.z;

    if (r_idx >= m || g_idx >= g || c_idx >= c) return;

    int q_idx = index_target[r_idx] * g * c + g_idx * c + c_idx;
    int k_idx = index_refer[r_idx] * g * c + g_idx * c + c_idx;
    int o_idx = r_idx * g + g_idx;
    float grad_r = grad_output[o_idx];
    atomicAdd(grad_query + q_idx, grad_r * key[k_idx] * weight[c_idx]);
    atomicAdd(grad_key + k_idx, grad_r * query[q_idx] * weight[c_idx]);
    atomicAdd(grad_weight + c_idx, grad_r * key[k_idx] * query[q_idx]);
}


__global__ void attention_fusion_step_forward_cuda_kernel(int m, int g, int c,
                                                          const float *weight, const float *value,
                                                          const int *index_target, const int *index_refer,
                                                          float *output)
{
    int r_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int g_idx = blockIdx.y;
    int c_idx = blockIdx.z;

    if (r_idx >= m || g_idx >= g || c_idx >= c) return;

    int o_idx = index_target[r_idx] * g * c + g_idx * c + c_idx;
    int v_idx = index_refer[r_idx] * g * c + g_idx * c + c_idx;

    float f = weight[r_idx * g + g_idx] * value[v_idx];
    atomicAdd(output + o_idx, f);
}


__global__ void attention_fusion_step_backward_cuda_kernel(int m, int g, int c,
                                                           const float *weight, float *grad_weight,
                                                           const float *value, float *grad_value,
                                                           const int *index_target, const int *index_refer,
                                                           const float *grad_output)
{
    int r_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int g_idx = blockIdx.y;
    int c_idx = blockIdx.z;

    if (r_idx >= m || g_idx >= g || c_idx >= c) return;

    int o_idx = index_target[r_idx] * g * c + g_idx * c + c_idx;
    int v_idx = index_refer[r_idx] * g * c + g_idx * c + c_idx;
    int w_idx = r_idx * g + g_idx;
    float grad = grad_output[o_idx];
    atomicAdd(grad_weight + w_idx, grad * value[v_idx]);
    atomicAdd(grad_value + v_idx, grad * weight[w_idx]);
}

/*
Launchers
*/


void attention_relation_step_forward_cuda_launcher(int m, int g, int c,
                                                   const float *query, const float *key, const float *weight,
                                                   const int *index_target, const int *index_refer,
                                                   float *output)
{
    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), g, c);
    dim3 threads(THREADS_PER_BLOCK);
    attention_relation_step_forward_cuda_kernel<<<blocks, threads, 0>>>(m, g, c, query, key, weight,
                                                                        index_target, index_refer, output);
}

void attention_relation_step_backward_cuda_launcher(int m, int g, int c,
                                                    const float *query, float *grad_query,
                                                    const float *key, float *grad_key,
                                                    const float *weight, float *grad_weight,
                                                    const int *index_target, const int *index_refer,
                                                    const float *grad_output)
{
    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), g, c);
    dim3 threads(THREADS_PER_BLOCK);
    attention_relation_step_backward_cuda_kernel<<<blocks, threads, 0>>>(m, g, c,
                                                                         query, grad_query,
                                                                         key, grad_key,
                                                                         weight, grad_weight,
                                                                         index_target, index_refer,
                                                                         grad_output);
}


void attention_fusion_step_forward_cuda_launcher(int m, int g, int c,
                                                 const float *weight, const float *value,
                                                 const int *index_target, const int *index_refer,
                                                 float *output)
{
    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), g, c);
    dim3 threads(THREADS_PER_BLOCK);
    attention_fusion_step_forward_cuda_kernel<<<blocks, threads, 0>>>(m, g, c, weight, value,
                                                                      index_target, index_refer, output);
}


void attention_fusion_step_backward_cuda_launcher(int m, int g, int c,
                                                  const float *weight, float *grad_weight,
                                                  const float *value, float *grad_value,
                                                  const int *index_target, const int *index_refer,
                                                  const float *grad_output)
{
    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), g, c);
    dim3 threads(THREADS_PER_BLOCK);
    attention_fusion_step_backward_cuda_kernel<<<blocks, threads, 0>>>(m, g, c,
                                                                       weight, grad_weight,
                                                                       value, grad_value,
                                                                       index_target, index_refer,
                                                                       grad_output);
}


