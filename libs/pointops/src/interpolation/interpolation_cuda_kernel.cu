#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "interpolation_cuda_kernel.h"


__global__ void interpolation_forward_cuda_kernel(int n, int c, int k, const float *input, const int *idx, const float *weight, float *output)
{
    // input: input: (m, c), idx: (n, k), weight: (n, k), output: output (n, c)
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n * c) return;
    int c_idx = index % c;
    int n_idx = index / c;
    for (int i = 0; i < k; i++)
    {
        int idx_idx = n_idx * k + i;
        int input_idx = idx[idx_idx] * c + c_idx;
        output[index] += input[input_idx] * weight[idx_idx];
    }
}

__global__ void interpolation_backward_cuda_kernel(int n, int c, int k, const float *grad_output, const int *idx, const float *weight, float *grad_input)
{
    // input: grad_output: (n, c), idx: (n, k), weight: (n, k), output: grad_input (m, c)
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n * c) return;
    int c_idx = index % c;
    int n_idx = index / c;
    for (int i = 0; i < k; i++)
    {
        int idx_idx = n_idx * k + i;
        int input_idx = idx[idx_idx] * c + c_idx;
        atomicAdd(grad_input + input_idx, grad_output[index] * weight[idx_idx]);
    }
}

void interpolation_forward_cuda_launcher(int n, int c, int k, const float *input, const int *idx, const float *weight, float *output) {
    // input: input: (m, c), idx: (n, k), weight: (n, k), output: output (n, c)
    dim3 blocks(DIVUP(n * c, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);
    interpolation_forward_cuda_kernel<<<blocks, threads, 0>>>(n, c, k, input, idx, weight, output);
}

void interpolation_backward_cuda_launcher(int n, int c, int k, const float *grad_output, const int *idx, const float *weight, float *grad_input) {
    // input: grad_output: (n, c), idx: (n, k), weight: (n, k), output: grad_input (m, c)
    dim3 blocks(DIVUP(n * c, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);
    interpolation_backward_cuda_kernel<<<blocks, threads, 0>>>(n, c, k, grad_output, idx, weight, grad_input);
}
