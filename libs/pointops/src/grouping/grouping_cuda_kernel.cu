#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "grouping_cuda_kernel.h"


__global__ void grouping_forward_cuda_kernel(int m, int nsample, int c, const float *__restrict__ input, const int *__restrict__ idx, float *__restrict__ output) {
    // input: input: (n, c), idx: (m, nsample), output: (m, nsample, c)
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= m * nsample * c) return;
    const int c_idx = index % c;
    const int nsample_idx = (index / c) % nsample;
    const int m_idx = index / nsample / c;
    const int input_idx = idx[m_idx * nsample + nsample_idx] * c + c_idx;
    output[index] = input[input_idx];
}

__global__ void grouping_backward_cuda_kernel(int m, int nsample, int c, const float *__restrict__ grad_output, const int *__restrict__ idx, float *__restrict__ grad_input) {
    // input: grad_output: (m, nsample, c), idx: (m, nsample), output: grad_input: (n, c)
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= m * nsample * c) return;
    const int c_idx = index % c;
    const int nsample_idx = (index / c) % nsample;
    const int m_idx = index / nsample / c;
    const int input_idx = idx[m_idx * nsample + nsample_idx] * c + c_idx;
    atomicAdd(grad_input + input_idx, grad_output[index]);
}

void grouping_forward_cuda_launcher(int m, int nsample, int c, const float *input, const int *idx, float *output) {
    // input: input: (n, c), idx: (m, nsample), output: (m, nsample, c)
    dim3 blocks(DIVUP(m * nsample * c, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);
    grouping_forward_cuda_kernel<<<blocks, threads, 0>>>(m, nsample, c, input, idx, output);
}

void grouping_backward_cuda_launcher(int m, int nsample, int c, const float *grad_output, const int *idx, float *grad_input)
{  
    // input: grad_output: (m, nsample, c), idx: (m, nsample), output: grad_input: (n, c)
    dim3 blocks(DIVUP(m * nsample * c, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);
    grouping_backward_cuda_kernel<<<blocks, threads, 0>>>(m, nsample, c, grad_output, idx, grad_input);
}
